#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>

#include "bicgstab_cuda.h"

#define CUDA_CHECK(err) do { \
    hipError_t _e = (err); \
    if (_e != hipSuccess) { \
        fprintf(stderr, "CUDA Error %s:%d: %s\n", \
                __FILE__, __LINE__, hipGetErrorString(_e)); \
        return; \
    } \
} while (0)

// Kernel: Q[row, n] = sum_col A[row, col] * P[col, n]
__global__ void batch_matvec_kernel(int dim, int batch,
                                    const double* __restrict__ mat,
                                    const double* __restrict__ P,
                                    double* __restrict__ Q) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < dim && n < batch) {
        double sum = 0.0;
        const double* Arow = mat + row * dim;
        const double* Pcol = P + n;
        for (int col = 0; col < dim; ++col) {
            sum += Arow[col] * Pcol[col * batch];
        }
        Q[row * batch + n] = sum;
    }
}

// Kernel: R = B - A * X
__global__ void batch_residual_kernel(int dim, int batch,
                                      const double* __restrict__ A,
                                      const double* __restrict__ X,
                                      const double* __restrict__ B,
                                      double* __restrict__ R) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < dim && n < batch) {
        double val = B[row * batch + n];
        const double* Arow = A + row * dim;
        const double* Xcol = X + n;
        for (int col = 0; col < dim; ++col) {
            val -= Arow[col] * Xcol[col * batch];
        }
        R[row * batch + n] = val;
    }
}

// Kernel: out[n] = dot( X[:,n], Y[:,n] )
__global__ void batch_dot_product_kernel(int dim, int batch,
                                         const double* __restrict__ X,
                                         const double* __restrict__ Y,
                                         double* __restrict__ out) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < batch) {
        double sum = 0.0;
        const double* Xcol = X + n;
        const double* Ycol = Y + n;
        for (int i = 0; i < dim; ++i) {
            sum += Xcol[i * batch] * Ycol[i * batch];
        }
        out[n] = sum;
    }
}

// Elementwise kernels
__global__ void batch_sqrt_kernel(int batch, const double* x, double* out) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < batch) out[n] = sqrt(x[n]);
}

__global__ void batch_mul_kernel(int batch, const double* x, const double* y, double* out) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < batch) out[n] = x[n] * y[n];
}

__global__ void batch_div_kernel(int batch, const double* x, const double* y, double* out) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < batch) out[n] = x[n] / y[n];
}

// Kernel: p = r + beta * (p - zeta * Ap)
__global__ void update_p_kernel(int dim, int batch,
                                double* __restrict__ out,
                                const double* __restrict__ r,
                                const double* __restrict__ p,
                                const double* __restrict__ Ap,
                                const double* __restrict__ beta,
                                const double* __restrict__ zeta) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < dim && n < batch) {
        int idx = row * batch + n;
        out[idx] = r[idx] + beta[n] * (p[idx] - zeta[n] * Ap[idx]);
    }
}

// Kernel: t = r - alpha * Akp
__global__ void update_t_kernel(int dim, int batch,
                                const double* __restrict__ r,
                                const double* __restrict__ Akp,
                                const double* __restrict__ alpha,
                                double* __restrict__ t) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < dim && n < batch) {
        int idx = row * batch + n;
        t[idx] = r[idx] - alpha[n] * Akp[idx];
    }
}

// Kernel: x += alpha * kp + zeta * kt
__global__ void update_x_kernel(int dim, int batch,
                                double* __restrict__ x,
                                const double* __restrict__ kp,
                                const double* __restrict__ kt,
                                const double* __restrict__ alpha,
                                const double* __restrict__ zeta) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < dim && n < batch) {
        int idx = row * batch + n;
        x[idx] += alpha[n] * kp[idx] + zeta[n] * kt[idx];
    }
}

// Kernel: r = t - zeta * Akt
__global__ void update_r_kernel(int dim, int batch,
                                const double* __restrict__ t,
                                const double* __restrict__ Akt,
                                const double* __restrict__ zeta,
                                double* __restrict__ r) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < dim && n < batch) {
        int idx = row * batch + n;
        r[idx] = t[idx] - zeta[n] * Akt[idx];
    }
}

void bicgstab_cuda(
    int batch,
    int dim,
    double** A,
    double** b,
    double** x,
    double tor,
    int max_steps
) {
    size_t mat_size = (size_t)dim * dim * sizeof(double);
    size_t batch_size = (size_t)dim * batch * sizeof(double);
    size_t vec_size = (size_t)batch * sizeof(double);

    // Device buffers
    double *d_A, *d_b, *d_x;
    CUDA_CHECK(hipMalloc(&d_A, mat_size));
    CUDA_CHECK(hipMalloc(&d_b, batch_size));
    CUDA_CHECK(hipMalloc(&d_x, batch_size));
    CUDA_CHECK(hipMemcpy(d_A, A[0], mat_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b[0], batch_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, x[0], batch_size, hipMemcpyHostToDevice));

    // Work arrays
    double *d_p, *d_r, *d_r0, *d_t;
    double *d_Ap, *d_Akp, *d_kt, *d_Akt, *d_kp;
    double *d_bnorm, *d_rnorm, *d_nom, *d_nom_old, *d_den;
    double *d_alpha, *d_beta, *d_zeta, *d_tmp;
    hipMalloc(&d_p, batch_size);
    hipMalloc(&d_r, batch_size);
    hipMalloc(&d_r0, batch_size);
    hipMalloc(&d_t, batch_size);
    hipMalloc(&d_Ap, batch_size);
    hipMalloc(&d_Akp, batch_size);
    hipMalloc(&d_kt, batch_size);
    hipMalloc(&d_Akt, batch_size);
    hipMalloc(&d_kp, batch_size);
    hipMalloc(&d_bnorm, vec_size);
    hipMalloc(&d_rnorm, vec_size);
    hipMalloc(&d_nom, vec_size);
    hipMalloc(&d_nom_old, vec_size);
    hipMalloc(&d_den, vec_size);
    hipMalloc(&d_alpha, vec_size);
    hipMalloc(&d_beta, vec_size);
    hipMalloc(&d_zeta, vec_size);
    hipMalloc(&d_tmp, vec_size);

    dim3 block2d(16, 16);
    dim3 grid2d((dim + 15) / 16, (batch + 15) / 16);
    int threads1d = 256;
    int blocks1d = (batch + threads1d - 1) / threads1d;

    // bnorm = sqrt(dot(b,b))
    batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_b, d_b, d_bnorm);
    batch_sqrt_kernel <<<blocks1d,threads1d>>>(batch, d_bnorm, d_bnorm);

    // r = b - A*x
    batch_residual_kernel<<<grid2d,block2d>>>(dim, batch, d_A, d_x, d_b, d_r);
    hipMemcpy(d_r0, d_r, batch_size, hipMemcpyDeviceToDevice);
    batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_r, d_r, d_rnorm);
    batch_sqrt_kernel <<<blocks1d,threads1d>>>(batch, d_rnorm, d_rnorm);

    double h_bnorm0, h_rnorm0;
    hipMemcpy(&h_bnorm0, d_bnorm + 0, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_rnorm0, d_rnorm + 0, sizeof(double), hipMemcpyDeviceToHost);
    printf("Original rel res [0] = %20.14e\n", h_rnorm0 / h_bnorm0);

    hipMemset(d_p, 0, batch_size);
    hipMemset(d_alpha, 0, vec_size);
    hipMemset(d_beta, 0, vec_size);
    hipMemset(d_zeta, 0, vec_size);

    // early exit
    batch_div_kernel<<<blocks1d,threads1d>>>(batch, d_rnorm, d_bnorm, d_tmp);
    hipMemcpy(&h_rnorm0, d_tmp + 0, sizeof(double), hipMemcpyDeviceToHost);
    if (h_rnorm0 < tor) goto finalize;

    for (int step = 1; step <= max_steps; ++step) {
        batch_matvec_kernel <<<grid2d,block2d>>>(dim, batch, d_A, d_p, d_Ap);
        update_p_kernel <<<grid2d,block2d>>>(dim, batch, d_p, d_r, d_p, d_Ap, d_beta, d_zeta);
        hipMemcpy(d_kp, d_p, batch_size, hipMemcpyDeviceToDevice);
        batch_matvec_kernel <<<grid2d,block2d>>>(dim, batch, d_A, d_kp, d_Akp);
        batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_r0, d_r, d_nom);
        batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_r0, d_Akp, d_den);
        batch_div_kernel <<<blocks1d,threads1d>>>(batch, d_nom, d_den, d_alpha);
        hipMemcpy(d_nom_old, d_nom, vec_size, hipMemcpyDeviceToDevice);

        update_t_kernel <<<grid2d,block2d>>>(dim, batch, d_r, d_Akp, d_alpha, d_t);
        hipMemcpy(d_kt, d_t, batch_size, hipMemcpyDeviceToDevice);
        batch_matvec_kernel <<<grid2d,block2d>>>(dim, batch, d_A, d_kt, d_Akt);
        batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_Akt, d_t, d_nom);
        batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_Akt, d_Akt, d_den);
        batch_div_kernel <<<blocks1d,threads1d>>>(batch, d_nom, d_den, d_zeta);

        update_x_kernel <<<grid2d,block2d>>>(dim, batch, d_x, d_kp, d_kt, d_alpha, d_zeta);
        update_r_kernel <<<grid2d,block2d>>>(dim, batch, d_t, d_Akt, d_zeta, d_r);

        // beta = alpha/zeta * dot(r0,r) / nom_old
        batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_r0, d_r, d_tmp);
        batch_mul_kernel <<<blocks1d,threads1d>>>(batch, d_alpha, d_tmp, d_beta);
        batch_div_kernel <<<blocks1d,threads1d>>>(batch, d_beta, d_zeta, d_beta);
        batch_div_kernel <<<blocks1d,threads1d>>>(batch, d_beta, d_nom_old, d_beta);

        // rnorm and check
        batch_dot_product_kernel<<<blocks1d,threads1d>>>(dim, batch, d_r, d_r, d_rnorm);
        batch_sqrt_kernel <<<blocks1d,threads1d>>>(batch, d_rnorm, d_rnorm);
        batch_div_kernel <<<blocks1d,threads1d>>>(batch, d_rnorm, d_bnorm, d_tmp);
        hipMemcpy(&h_rnorm0, d_tmp + 0, sizeof(double), hipMemcpyDeviceToHost);
        printf("  Step %d rel res [0] = %20.14e\n", step, h_rnorm0);
        if (h_rnorm0 < tor) break;
    }

finalize:
    hipMemcpy(x[0], d_x, batch_size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_r0);
    hipFree(d_t);
    hipFree(d_Ap);
    hipFree(d_Akp);
    hipFree(d_kt);
    hipFree(d_Akt);
    hipFree(d_kp);
    hipFree(d_bnorm);
    hipFree(d_rnorm);
    hipFree(d_nom);
    hipFree(d_nom_old);
    hipFree(d_den);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_zeta);
    hipFree(d_tmp);
}
